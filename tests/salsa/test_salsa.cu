#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_salsa.cu
 *
 * @brief Simple test driver program for using SALSA algorithm to compute rank.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <iostream>
#include <cstdlib>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BFS includes
#include <gunrock/app/salsa/salsa_enactor.cuh>
#include <gunrock/app/salsa/salsa_problem.cuh>
#include <gunrock/app/salsa/salsa_functor.cuh>

// Operator includes
#include <gunrock/oprtr/edge_map_forward/kernel.cuh>
#include <gunrock/oprtr/vertex_map/kernel.cuh>

#include <moderngpu.cuh>

// boost includes
#include <boost/config.hpp>
#include <boost/utility.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/page_rank.hpp>


using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::salsa;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

template <typename VertexId, typename Value>
struct RankPair {
    VertexId        vertex_id;
    Value           page_rank;

    RankPair(VertexId vertex_id, Value page_rank) : vertex_id(vertex_id), page_rank(page_rank) {}
};

template<typename RankPair>
bool SALSACompare(
    RankPair elem1,
    RankPair elem2)
{
    return elem1.page_rank > elem2.page_rank;
}

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
 printf("\ntest_salsa <graph type> <graph type args> [--device=<device_index>] "
        "[--undirected] [--instrumented] [--quick] "
        "[--v]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --undirected If set then treat the graph as undirected.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --quick If set will skip the CPU validation code.\n"
        );
 }

 /**
  * @brief Displays the BFS result (i.e., distance from source)
  *
  * @param[in] source_path Search depth from the source for each node.
  * @param[in] nodes Number of nodes in the graph.
  */
 template<typename Value, typename SizeT>
 void DisplaySolution(Value *hrank, Value *arank, SizeT nodes)
 { 
     //sort the top page ranks
     RankPair<SizeT, Value> *hr_list = (RankPair<SizeT, Value>*)malloc(sizeof(RankPair<SizeT, Value>) * nodes);
     RankPair<SizeT, Value> *ar_list = (RankPair<SizeT, Value>*)malloc(sizeof(RankPair<SizeT, Value>) * nodes);

     for (int i = 0; i < nodes; ++i)
     {
         hr_list[i].vertex_id = i;
         hr_list[i].page_rank = hrank[i];
         ar_list[i].vertex_id = i;
         ar_list[i].page_rank = arank[i];
     }
     std::stable_sort(hr_list, hr_list + nodes, SALSACompare<RankPair<SizeT, Value> >);
     std::stable_sort(ar_list, ar_list + nodes, SALSACompare<RankPair<SizeT, Value> >);

     // Print out at most top 10 largest components
     int top = (nodes < 10) ? nodes : 10;
     printf("Top %d Page Ranks:\n", top);
     for (int i = 0; i < top; ++i)
     {
         printf("Vertex ID: %d, Hub Rank: %5f\n", hr_list[i].vertex_id, hr_list[i].page_rank);
         printf("Vertex ID: %d, Authority Rank: %5f\n", ar_list[i].vertex_id, ar_list[i].page_rank);
     }

     free(hr_list);
     free(ar_list);
 }

 /**
  * Performance/Evaluation statistics
  */ 

struct Stats {
    char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * 
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] h_rank Host-side vector stores computed page rank values for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] total_queued Total element queued in BFS kernel running process
 * @param[in] avg_duty Average duty of the BFS kernels
 */

void DisplayStats(
    Stats               &stats,
    double              elapsed,
    double              avg_duty)
{
    
    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display the specific sample statistics
    printf(" elapsed: %.3f ms", elapsed);
    if (avg_duty != 0) {
        printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
    }
    printf("\n");
}




/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

 /**
  * @brief A simple CPU-based reference Page Rank implementation.
  *
  * @tparam VertexId
  * @tparam Value
  * @tparam SizeT
  *
  * @param[in] graph Reference to the CSR graph we process on
  * @param[in] rank Host-side vector to store CPU computed labels for each node
  * @param[in] delta delta for computing SALSA rank
  * @param[in] error error threshold
  * @param[in] max_iter max iteration to go
  */
 template<
    typename VertexId,
    typename Value,
    typename SizeT>
void SimpleReferenceSALSA(
    const Csr<VertexId, Value, SizeT>       &graph,
    const Csr<VertexId, Value, SizeT>       &inv_graph,
    Value                                   *hrank,
    Value                                   *arank,
    SizeT                                   max_iter) 
{
    using namespace boost;

    //Preparation
    
    //
    //compute SALSA rank
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    printf("CPU BFS finished in %lf msec.\n", elapsed);
}

/**
 * @brief Run SALSA tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] delta Delta value for computing PageRank, usually set to .85
 * @param[in] error Error threshold value
 * @param[in] max_iter Max iteration for Page Rank computing
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] context CudaContext for moderngpu to use
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT>
void RunTests(
    const Csr<VertexId, Value, SizeT> &graph,
    const Csr<VertexId, Value, SizeT> &inv_graph,
    SizeT max_iter,
    int max_grid_size,
    int num_gpus,
    CudaContext& context)
{
    
    typedef SALSAProblem<
        VertexId,
        SizeT,
        Value> Problem;

        // Allocate host-side label array (for both reference and gpu-computed results)
        Value    *reference_hrank       = (Value*)malloc(sizeof(Value) * graph.nodes);
        Value    *reference_arank       = (Value*)malloc(sizeof(Value) * graph.nodes);
        Value    *h_hrank               = (Value*)malloc(sizeof(Value) * graph.nodes);
        Value    *h_arank               = (Value*)malloc(sizeof(Value) * graph.nodes);
        Value    *reference_check_h     = (g_quick) ? NULL : reference_hrank;
        Value    *reference_check_a     = (g_quick) ? NULL : reference_arank;

        // Allocate BFS enactor map
        SALSAEnactor<INSTRUMENT> salsa_enactor(g_verbose);

        // Allocate problem on GPU
        Problem *csr_problem = new Problem;
        util::GRError(csr_problem->Init(
            g_stream_from_host,
            graph,
            inv_graph,
            num_gpus), "Problem SALSA Initialization Failed", __FILE__, __LINE__);

        //
        // Compute reference CPU SALSA solution for source-distance
        //
        if (reference_check_h != NULL)
        {
            printf("compute ref value\n");
            SimpleReferenceSALSA(
                    graph,
                    inv_graph,
                    reference_check_h,
                    reference_check_a,
                    max_iter);
            printf("\n");
        }

        Stats *stats = new Stats("GPU SALSA");

        long long           total_queued = 0;
        double              avg_duty = 0.0;

        // Perform BFS
        GpuTimer gpu_timer;

        util::GRError(csr_problem->Reset(salsa_enactor.GetFrontierType()), "SALSA Problem Data Reset Failed", __FILE__, __LINE__);
        gpu_timer.Start();
        util::GRError(salsa_enactor.template Enact<Problem>(context, csr_problem, max_iter, max_grid_size), "SALSA Problem Enact Failed", __FILE__, __LINE__);
        gpu_timer.Stop();

        salsa_enactor.GetStatistics(total_queued, avg_duty);

        double elapsed = gpu_timer.ElapsedMillis();

        // Copy out results
        util::GRError(csr_problem->Extract(h_hrank, h_arank), "SALSA Problem Data Extraction Failed", __FILE__, __LINE__);

        // Verify the result
        if (reference_check_a != NULL) {
            printf("Validity: ");
            CompareResults(h_hrank, reference_check_h, graph.nodes, true);
            CompareResults(h_arank, reference_check_a, graph.nodes, true);
        }
        printf("\nFirst 40 labels of the GPU result."); 
        // Display Solution
        DisplaySolution(h_hrank, h_arank, graph.nodes);

        DisplayStats(
            *stats,
            elapsed,
            avg_duty);


        // Cleanup
        delete stats;
        if (csr_problem) delete csr_problem;
        if (reference_check_h) free(reference_check_h);
        if (reference_check_a) free(reference_check_a);

        if (h_hrank) free(h_hrank);
        if (h_arank) free(h_arank);

        hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    Csr<VertexId, Value, SizeT> &inv_graph,
    CommandLineArgs &args,
    CudaContext& context)
{
    SizeT               max_iter            = 20;
    bool                instrumented        = false;        // Whether or not to collect instrumentation from kernels
    int                 max_grid_size       = 0;            // maximum grid size (0: leave it up to the enactor)
    int                 num_gpus            = 1;            // Number of GPUs for multi-gpu enactor to use

    instrumented = args.CheckCmdLineFlag("instrumented");
    args.GetCmdLineArgument("max-iter", max_iter);

    g_quick = args.CheckCmdLineFlag("quick");
    g_verbose = args.CheckCmdLineFlag("v");

    if (instrumented) {
        RunTests<VertexId, Value, SizeT, true>(
                        graph,
                        inv_graph,
                        max_iter,
                        max_grid_size,
                        num_gpus,
                        context);
    } else {
        RunTests<VertexId, Value, SizeT, false>(
                        graph,
                        inv_graph,
                        max_iter,
                        max_grid_size,
                        num_gpus,
                        context);
    }
}



/******************************************************************************
* Main
******************************************************************************/

int main( int argc, char** argv)
{
	CommandLineArgs args(argc, argv);

	if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
		Usage();
		return 1;
	}

	//DeviceInit(args);
	//hipSetDeviceFlags(hipDeviceMapHost);
	int dev = 0;
    args.GetCmdLineArgument("device", dev);
    ContextPtr context = mgpu::CreateCudaDevice(dev);

	//srand(0);									// Presently deterministic
	//srand(time(NULL));

	// Parse graph-contruction params
	g_undirected = false;

	std::string graph_type = argv[1];
	int flags = args.ParsedArgc();
	int graph_args = argc - flags - 1;

	if (graph_args < 1) {
		Usage();
		return 1;
	}
	
	//
	// Construct graph and perform search(es)
	//

	if (graph_type == "market") {

		// Matrix-market coordinate-formatted graph file

		typedef int VertexId;							// Use as the node identifier type
		typedef float Value;								// Use as the value type
		typedef int SizeT;								// Use as the graph size type
		Csr<VertexId, Value, SizeT> csr(false);         // default value for stream_from_host is false

		Csr<VertexId, Value, SizeT> inv_csr(false);

		if (graph_args < 1) { Usage(); return 1; }
		char *market_filename = (graph_args == 2) ? argv[2] : NULL;
		if (graphio::BuildMarketGraph<false>(
			market_filename, 
			csr, 
			g_undirected,
			false) != 0) 
		{
			return 1;
		}

        if (graphio::BuildMarketGraph<false>(
                    market_filename, 
                    inv_csr, 
                    g_undirected,
                    true) != 0) 
        {
            return 1;
        }

		csr.PrintHistogram();

		    // Run tests
		    RunTests(csr, inv_csr, args, *context);

	} else {

		// Unknown graph type
		fprintf(stderr, "Unspecified graph type\n");
		return 1;

	}

	return 0;
} 
